#include <hip/hip_runtime.h>

#include <cctype>

#include <inttypes.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <iostream>
#include <algorithm>

unsigned long get_elapsed(struct timespec *start, struct timespec *end)
{
    uint64_t dur;
    dur = ((uint64_t)end->tv_sec * 1000000000 + end->tv_nsec) - 
        ((uint64_t)start->tv_sec * 1000000000 + start->tv_nsec);
    return (unsigned long)dur;
}

std::string processCipher( std::string input, std::string key, bool encode )
{
    struct timespec tstart, tend;

    clock_gettime(CLOCK_REALTIME, &tstart);

    std::string output = input;

    input.erase( remove_if(input.begin(), input.end(), isspace), input.end() );
    std::transform(input.begin(), input.end(), input.begin(), ::tolower);
    std::transform(key.begin(), key.end(), key.begin(), ::tolower);

    for(int i=0; i<input.length(); i++)
    {
        int keyValue = (int)key[i%key.length()] - 97;
        int textValue = (int)input[i] - 97;

        if(encode)
            output[i] = (char)( ( ( textValue + keyValue ) % 26 ) + 97 );
        else
            output[i] = (char)( ( ( textValue + ( 26 - keyValue ) ) % 26 ) + 97 );
    }

    clock_gettime(CLOCK_REALTIME, &tend);

    printf("Cipher Processing: %ld usec\n", get_elapsed(&tstart, &tend)/1000);

    return output;
}

void runCipher(bool encode)
{
    std::string input, output;
    std::string key;

    if(encode)
        std::cout << "Please input the text you wish to encode:" << std::endl;
    else
        std::cout << "Please input the text you wish to decode:" << std::endl;

    std::getline( std::cin, input );

    std::cout << "Please input the key:" << std::endl;
    std::getline( std::cin, key );

    std::cout << "Your cipher text is:" << std::endl;

    output = processCipher(input, key, encode);

    std::cout << output << std::endl;
}

int main(int argc, char **argv)
{
    std::string choice = "encode";
    bool loop = true;

    std::cout<<"Would you like to encode or decode?"<<std::endl;
    std::getline( std::cin, choice );

    while(loop)
    {
        if( choice == "encode")
        {
            runCipher(true);
            loop = false;
        }
        else if( choice == "decode")
        {
            runCipher(false);
            loop = false;
        }
        else
        {
            std::cout << "That is not a valid input, please input either 'encode' or 'decode'" << std::endl;
            std::getline( std::cin, choice );
        }
    }

    return 0;
}

